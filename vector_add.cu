#include <stdio.h>
#include <hip/hip_runtime.h>

#define SIZE 1024*1024*32  // Define the size of the vectors

__global__ void vectorAdd(int *A, int *B, int *C, int n) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}

void verifyResult(int *C) {
    for (int i = 0; i < SIZE; i++) {
        if (C[i] != SIZE) {  // Each element should be SIZE - 1
            printf("Error at index %d: expected %d, got %d\n", i, SIZE - 1, C[i]);
            return;
        }
    }
    printf("All results are correct!\n");
}

int main() {
    int *A, *B, *C;
    int *d_A, *d_B, *d_C;
    int size = SIZE * sizeof(int);

    // CUDA event creation, used for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    A = (int *)malloc(size);
    B = (int *)malloc(size);
    C = (int *)malloc(size);

    for(int i = 0; i < SIZE; i++) {
        A[i] = i;
        B[i] = SIZE - i;
    }

    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // Start recording
    hipEventRecord(start);

    int numThreadsPerBlock = 256;
    int blocksPerGrid = (SIZE + numThreadsPerBlock - 1) / numThreadsPerBlock;
    vectorAdd<<<blocksPerGrid, numThreadsPerBlock>>>(d_A, d_B, d_C, SIZE);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    // Stop recording
    hipEventRecord(stop);

    // Calculate and print the execution time
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Execution time on GPU: %f milliseconds\n", milliseconds);

    verifyResult(C);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(A);
    free(B);
    free(C);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}