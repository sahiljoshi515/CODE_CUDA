#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cassert>

# define INPUT 4096
# define KERNEL 1024
# define OUTPUT (INPUT - KERNEL + 1)

__global__ void conv1d(int *input, int *kernel, int *output) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate the index of the output element
    if (idx < OUTPUT) { // Ensure we don't go out of bounds
        int sum = 0;
        for (int j = 0; j < KERNEL; j++) {
            sum += input[idx + j] * kernel[j]; // Perform the convolution operation
        }
        output[idx] = sum; // Store the result in output
    }
}


// Function to initialize the colors in the image
void initializeConvolution(int *input, int *kernel, int *output) {
    for (int i = 0; i < INPUT; i++) {
        input[i] = 1.0; // Initialize input with some values
    }
    for (int i = 0; i < KERNEL; i++) {
        kernel[i] = 2.0; // Initialize kernel with some values 
    }
    for (int i = 0; i < INPUT - KERNEL + 1; i++) {
        output[i] = 0.0f; // Initialize output to zero
    }
}

void verifyResult(int *input, int *kernel, int *output) {
    for (int i = 0; i < OUTPUT; i++) {
        int expected = 0;
        for (int j = 0; j < KERNEL; j++) {
            expected += input[i + j] * kernel[j]; // Perform the convolution operation
        }
        assert(output[i] == expected); // Check the result
    }
    printf("All results are correct!\n");
}

int main() {
    int *input, *kernel, *output;
    int *d_input, *d_kernel, *d_output;

    size_t input_size = INPUT * sizeof(int);
    size_t kernel_size = KERNEL * sizeof(int);
    size_t output_size = OUTPUT * sizeof(int);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    input = (int *)malloc(input_size);
    kernel = (int *)malloc(kernel_size);
    output = (int *)malloc(output_size);
    hipMalloc((void **)&d_input, input_size);
    hipMalloc((void **)&d_kernel, kernel_size);
    hipMalloc((void **)&d_output, output_size);

    initializeConvolution(input, kernel, output);

    hipMemcpy(d_input, input, input_size, hipMemcpyHostToDevice); 
    hipMemcpy(d_kernel, kernel, kernel_size, hipMemcpyHostToDevice); 

    int threadsPerBlock = 256; // Number of threads per block
    int blocksPerGrid = (OUTPUT + threadsPerBlock - 1) / threadsPerBlock; // Calculate the number of blocks needed

    hipEventRecord(start);

    conv1d<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_kernel, d_output);

    hipMemcpy(output, d_output, output_size, hipMemcpyDeviceToHost);

    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Execution time on GPU: %f milliseconds\n", milliseconds);

    verifyResult(input, kernel, output);

    hipFree(d_input);
    hipFree(d_kernel);
    hipFree(d_output);
    free(input);
    free(kernel);
    free(output);

    hipEventDestroy(start);
    hipEventDestroy(stop);
}